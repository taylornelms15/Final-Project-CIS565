#include "hip/hip_runtime.h"
/**
@file pointMatcher.cu
@author Taylor Nelms
*/





#include "pointMatcher.h"

float* d_A1;
float* d_A2;
float* d_O;



__global__ void multiplyNumbers(float* A1, float* A2, float* O){
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    float intermediary = A1[index] * A2[index];
    O[index] = intermediary;


}//multiplyNumbers



float testCudaFunctionality(float* arrayOne, float* arrayTwo){

    float O[32];

    hipMalloc(&d_A1, 32);
    hipMalloc(&d_A2, 32);
    hipMalloc(&d_O, 32);

    dim3 threadsPerBlock(32);
    dim3 blocksPerGrid(1);


    hipMemcpy(d_A1, arrayOne, 32 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_A2, arrayTwo, 32 * sizeof(float), hipMemcpyHostToDevice);

    multiplyNumbers<<< blocksPerGrid, threadsPerBlock >>>(d_A1, d_A2, O);

    hipMemcpy(O, d_O, 32 * sizeof(float), hipMemcpyDeviceToHost);

    float result = 0.0f;
    for (int i = 0; i < 32; i++){
        result += O[i]; 
    }//for

    hipFree(d_A1);
    hipFree(d_A1);
    hipFree(d_O);

    return result;
    


}//testCudaFunctionality
