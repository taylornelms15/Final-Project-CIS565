#include "hip/hip_runtime.h"
/**
@file pointMatcher.cu
@author Taylor Nelms
*/





#include "pointMatcher.h"


void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": line %d: %s: %s\n", line, msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}


#define MINTIME 0.001f
#define DIST_THRESH 0.02f

float* d_A1;
float* d_A2;
float* d_O;

#define DOTP(a,b) (a.x * b.x + a.y * b.y + a.z * b.z)

/**
Ray 1: origin a, direction b
Ray 2: origin c, direction d
@return[0:2]: midpoint of where they come closest
@return[3]: their closest distance from each other
*/
__host__ __device__ glm::vec4 closestRayIntersect(glm::vec3 a, glm::vec3 b, glm::vec3 c, glm::vec3 d){
    //float s = DOTP(b, d) * (DOTP(a, d) - DOTP(b, c)) - DOTP(a, d) * DOTP(c, d) / (DOTP(b, d) * DOTP(b, d) - 1);
    //float t = DOTP(b, d) * (DOTP(c, d) - DOTP(a, d)) - DOTP(b, c) * DOTP(a, b) / (DOTP(b, d) * DOTP(b, d) - 1);
    glm::vec3 ab = c - a;
    float s = (-DOTP(b, d) * DOTP(d, ab) + DOTP(b, ab) * DOTP(d, d)) / (DOTP(b, b) * DOTP(d, d) - DOTP(b, d) * DOTP(b, d));
    float t = (DOTP(b, d) * DOTP(b, ab) - DOTP(d, ab) * DOTP(b, b)) / (DOTP(b, b) * DOTP(d, d) - DOTP(b, d) * DOTP(b, d));

    //glm::vec3 closest1 = a + b * t;
    //glm::vec3 closest2 = c + d * s;
    glm::vec3 closest1 = a + b * s;
    glm::vec3 closest2 = c + d * t;
    float dist = glm::distance(closest1, closest2);
    if (s < MINTIME || t < MINTIME)
        dist = 1000000.0f;//don't accept negative points on the ray!
    glm::vec3 midpoint = 0.5f * (closest1 + closest2);

    return glm::vec4(midpoint, dist);

}//closestRayIntersect

__global__ void multiplyNumbers(float* A1, float* A2, float* O){
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    float intermediary = A1[index] * A2[index];
    O[index] = intermediary;


}//multiplyNumbers

/**
This converts from a space where a positive Z value is "forward from the camera"
into the space that the rotation wants for our view vector
*/
tf2::Vector3 zForwardToOrientation(float X, float Y, float Z){
    //tf2::Vector3 Da1 = tf2::Vector3(X1, Y1, Z1);
    //camera moving right, we look towards the -x direction as our Z increases
    //So, for our construction space, o1=(0,0,0), o2=(1,0,0)
    //and "forward" for both is around (0,0,1), with "up" at (0,1,0) and "right" at (1,0,0)
    //we rotate to a world where o1=(0,0,0), o2=(0,0,1)
    //and "forward" for both is (-1,0,0), with "up" at (0,-1,0) and "right" at (0,0,1)
    //-Z, X, Y
    //return tf2::Vector3(-Z, X, Y);
    //return tf2::Vector3(Z, X, Y);
    return tf2::Vector3(X, Y, Z);

}//zForwardToOrientation

PointSub matchTwoPoints(PointSub pt1, 
                        tf2::Transform xform1, 
                        PointSub pt2, 
                        tf2::Transform xform2,
                        float fov,
                        float* distance,//out param
                        int width, int height){
    //make our rotation-only transforms                    
    tf2::Transform rotate1 = tf2::Transform(xform1.getRotation());
    tf2::Transform rotate2 = tf2::Transform(xform2.getRotation());
    //make return value
    PointSub retval = {};
    //get NDC ranging [0:1]
    float percentX1 = pt1.x / width; float percentY1 = pt1.y / height;
    float percentX2 = pt2.x / width; float percentY2 = pt2.y / height;
    //make the range go from -1 to 1
    percentX1 = (percentX1 - 0.5f) * 2.0f; percentY1 = (percentY1 - 0.5f) * 2.0f;
    percentX2 = (percentX2 - 0.5f) * 2.0f; percentY2 = (percentY2 - 0.5f) * 2.0f;
    float aspectRatio = (width * 1.0f) / height;
    percentX1 *= aspectRatio; percentX2 *= aspectRatio;
    //turn these into ray pieces
    float halftan = glm::tan(glm::radians(fov / 2.0f));
    float Xa1 = percentX1 * halftan; float Ya1 = percentY1 * halftan;
    float Xa2 = percentX2 * halftan; float Ya2 = percentY2 * halftan;
    //floats to go into vectors
    float X1 = Xa1; float Y1 = Ya1; float Z1 = 1.0f;
    float X2 = Xa2; float Y2 = Ya2; float Z2 = 1.0f;
    //make direction vector and transform
    tf2::Vector3 Da1 = zForwardToOrientation(X1, Y1, Z1);
    tf2::Vector3 Da2 = zForwardToOrientation(X2, Y2, Z2);
    tf2::Vector3 D1 = rotate1(Da1);
    tf2::Vector3 D2 = rotate2(Da2);
    //make glm vectors for position, direction
    tf2::Vector3 o1 = xform1.getOrigin();
    tf2::Vector3 o2 = xform2.getOrigin();
    glm::vec3 p1 = glm::vec3(o1[0], o1[1], o1[2]);
    glm::vec3 p2 = glm::vec3(o2[0], o2[1], o2[2]);
    glm::vec3 d1 = glm::normalize(glm::vec3(D1[0], D1[1], D1[2]));
    glm::vec3 d2 = glm::normalize(glm::vec3(D2[0], D2[1], D2[2]));
    //get matching point and distance
    glm::vec4 match = closestRayIntersect(p1, d1, p2, d2);
    //fill in our return values
    *distance = match.w;
    retval.x = match.x; retval.y = match.y; retval.z = match.z;
    //average the colors
    retval.r = pt1.r + pt2.r / 2;
    retval.g = pt1.g + pt2.g / 2;
    retval.b = pt1.b + pt2.b / 2;

    //FAKE: return a "view ray direction" position
    //tf2::Vector3 viewDirec = D1 * 0.3f;
    //tf2::Vector3 viewDireca = o1 + viewDirec;
    //retval.x = viewDireca[0]; retval.y = viewDireca[1]; retval.z = viewDireca[2];
    //retval.r = pt1.r; retval.g = pt1.g; retval.b = pt1.b;


    return retval;
}//matchTwoPoints


float testCudaFunctionality(float* arrayOne, float* arrayTwo){

    float O[32];

    hipMalloc(&d_A1, 32 * sizeof(float));
    hipMalloc(&d_A2, 32 * sizeof(float));
    hipMalloc(&d_O, 32 * sizeof(float));
    checkCUDAError("cudaMallocs error");

    dim3 threadsPerBlock(32);
    dim3 blocksPerGrid(1);


    hipMemcpy(d_A1, arrayOne, 32 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_A2, arrayTwo, 32 * sizeof(float), hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy error");

    multiplyNumbers<<< blocksPerGrid, threadsPerBlock >>>(d_A1, d_A2, d_O);

    hipMemcpy(O, d_O, 32 * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    float result = 0.0f;
    for (int i = 0; i < 32; i++){
        result += O[i]; 
    }//for

    hipFree(d_A1);
    hipFree(d_A1);
    hipFree(d_O);

    return result;
    


}//testCudaFunctionality

/**
Converts an (x,y) coordinate to the relevant RGB value in the image
*/
__host__ Vec3b coordToColor(Mat img, Point2f coord){
    return img.at<Vec3b>(coord);//is it this easy?
    float col = coord.x;
    float row = coord.y;//img.rows - coord.y;//top-down vs bottom-up conversion
    return img.at<Vec3b>((int)(row + 0.5f), (int)(col + 0.5));

}//coordToColor

void getCameraIntrinsicMatrix(
        Mat                     img1,
        std::vector<KeyPoint>   keypoints1,
        tf2::Transform          xform1,
        Mat                     img2,
        std::vector<KeyPoint>   keypoints2,
        tf2::Transform          xform2,
        std::vector<DMatch>     good_matches,
        Mat*                    output
){
    Mat retval = Mat();
    tf2::Vector3 o1 = xform1.getOrigin();
    tf2::Vector3 o2 = xform2.getOrigin();
    tf2::Quaternion r1 = xform1.getRotation();
    tf2::Quaternion r2 = xform2.getRotation();
    std::vector<Point2f> img1Points = std::vector<Point2f>();
    std::vector<Point2f> img2Points = std::vector<Point2f>();
    printf("==MATCHES==\n");
    for (int i = 0; i < good_matches.size(); i++){
        KeyPoint img1pt = keypoints1[good_matches[i].trainIdx];
        KeyPoint img2pt = keypoints2[good_matches[i].queryIdx];
        img1Points.push_back(img1pt.pt);
        img2Points.push_back(img2pt.pt);
    }
    Mat fundamental = findFundamentalMat(img1Points, img2Points, FM_RANSAC);
    Mat H1, H2;
    stereoRectifyUncalibrated(img1Points, img2Points, fundamental, Size(img1.cols, img1.rows), H1, H2);


    *output = Mat(retval);
}

std::vector<PointSub> getMatchingWorldPointsAlt(
        Mat                     img1,
        std::vector<KeyPoint>   keypoints1,
        tf2::Transform          xform1,
        Mat                     img2,
        std::vector<KeyPoint>   keypoints2,
        tf2::Transform          xform2,
        std::vector<DMatch>     good_matches,
        float                   FoV)
{
    std::vector<PointSub> retval = std::vector<PointSub>();
    std::vector<Point2f> img1Points = std::vector<Point2f>();
    std::vector<Point2f> img2Points = std::vector<Point2f>();
    printf("==MATCHES==\n");
    for (int i = 0; i < good_matches.size(); i++){
        KeyPoint img1pt = keypoints1[good_matches[i].trainIdx];
        KeyPoint img2pt = keypoints2[good_matches[i].queryIdx];
        img1Points.push_back(img1pt.pt);
        img2Points.push_back(img2pt.pt);
    }
    Mat fundamental = findFundamentalMat(img1Points, img2Points, FM_RANSAC);
    Mat H1, H2;
    stereoRectifyUncalibrated(img1Points, img2Points, fundamental, Size(img1.cols, img1.rows), H1, H2);
    //std::vector<Vec3f> lines1, lines2;
    //computeCorrespondEpilines(img1Points, 1, fundamental, lines1);
    //computeCorrespondEpilines(img2Points, 2, fundamental, lines2);


    return retval;
}//getMathingWorldPointsAlt

std::vector<PointSub> getMatchingWorldPoints(
        Mat                     img1,
        std::vector<KeyPoint>   keypoints1,
        tf2::Transform          xform1,
        Mat                     img2,
        std::vector<KeyPoint>   keypoints2,
        tf2::Transform          xform2,
        std::vector<DMatch>     good_matches,
        float                   FoV)
{
    std::vector<PointSub> retval = std::vector<PointSub>();
    //Keypoints: given in (x, y) coordinates (scaled as pixels, origin bottom left (likely)
    //good_matches: given in (query, train) pairs: indices of the keypoints1 and keypoints2 entries

    std::vector<PointSub> img1Points = std::vector<PointSub>();//not the actual world points
    std::vector<PointSub> img2Points = std::vector<PointSub>();//not the actual world points
    printf("==MATCHES==\n");
    for (int i = 0; i < good_matches.size(); i++){
        KeyPoint img1pt = keypoints1[good_matches[i].trainIdx];
        KeyPoint img2pt = keypoints2[good_matches[i].queryIdx];
        PointSub pt1, pt2;
        pt1.x = img1pt.pt.x; pt1.y = /*img1.rows -*/ img1pt.pt.y;
        pt2.x = img2pt.pt.x; pt2.y = /*img1.rows -*/ img2pt.pt.y;
        Vec3b col1 = coordToColor(img1, img1pt.pt);
        Vec3b col2 = coordToColor(img2, img2pt.pt);
        pt1.b = col1[0]; pt1.g = col1[1]; pt1.r = col1[2];
        pt2.b = col2[0]; pt2.g = col2[1]; pt2.r = col2[2];
        img1Points.push_back(pt1);
        img2Points.push_back(pt2);
    }

    int width = img1.cols;
    int height = img1.rows;

    float totalDist = 0.0f;

    for(int i = 0; i < img1Points.size(); i++){
        float distance;
        PointSub resultMatch = matchTwoPoints(img1Points.at(i), xform1,
                                              img2Points.at(i), xform2,
                                              FoV,
                                              &distance,
                                              width, height);
        if (distance < DIST_THRESH){
            retval.push_back(resultMatch); 
            totalDist += distance;
        }
    }//for

    printf("\tAvg distance %f\n", totalDist / retval.size());



    return retval;
}//getMatchingWorldPoints














